#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <io.h>
#include <hiprand/hiprand_kernel.h>

#include "geo_list.cuh"
#include "sphere.cuh"
#include "cam.cuh"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

#define RANDVEC3 vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))

__device__ 
vec3 random_in_unit_sphere(hiprandState* local_rand_state) 
{
    vec3 p;
    do {
        p = 2.0f * RANDVEC3 - vec3(1, 1, 1);
    } while (p.squared_length() >= 1.0f);
    return p;
}

__device__ 
vec3 colour(const ray& r, geometry** world, hiprandState* local_rand_state) 
{
    ray cur_ray = r;
    float cur_attenuation = 1.0f;
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
            cur_attenuation *= 0.5f;
            cur_ray = ray(rec.p, target - rec.p);
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ 
void render_init(int max_x, int max_y, hiprandState* rand_state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ 
void render(vec3* fb, int max_x, int max_y, int ns, camera** cam, 
    geometry** world, hiprandState* rand_state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += colour(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

__global__
void create_world(geometry** d_list, geometry** d_world, camera** d_camera, float a) 
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new geo_list(d_list, 2);
        *d_camera = new camera(a);
    }
}

__global__ 
void free_world(geometry** d_list, geometry** d_world, camera** d_camera) 
{
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
    delete* d_camera;
}

int main() 
{
    //480p
    int width = 640;
    int height = 480;
    float aspect_ratio = (float)width / height;
    //Recursion depth of 25 (i.e. 25 ray bounces)
    int depth = 25;

    /*
    My RTX 2060 has:
    - 30 SMs
    - 64 CUDA cores per SM
    - supports 1024 threads per SM (i.e. 1024 threads per block)
    - warp = how threads are packaged: into packs of 32, block are split into warps
    -->best practice, i.e. maximum efficiency, have block sizes be multiples of warp size (32*x)
    - N / blocksize = number of blocks * blocksize ensures I have N threads of execution.
    - (N + blockSize - 1) / blockSize ensures I create the correct number of blocks for when N isn't a mutiple of blockSize.
    - N = 640*480 = 9600 * 32
    --> Make each block the max size of 1024 (32 warps), 300 blocks of 32 warps maximizes GPU usage
    */
    int bx = 16;
    int by = 16;

    int N = width * height;
    size_t fb_size = 3 * N * sizeof(vec3);

    // allocate frameBuffer
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, N * sizeof(hiprandState)));

    // Camera setup
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    
    //World setup
    geometry** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(geometry*)));
    geometry** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(geometry*)));
    
    create_world<<<1, 1>>>(d_list, d_world, d_camera, aspect_ratio);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // Render our buffer
    dim3 blocks(width / bx, height / by);
    dim3 threads(bx, by);
    render_init<<<blocks, threads>>>(width, height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb, width, height, depth, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    // Output FB as Image
    FILE* fd;
    if (fopen_s(&fd, "out/test_render_4.ppm", "w") != 0 || fd == NULL)
        exit(errno);

    if (_dup2(_fileno(fd), _fileno(stdout)) != 0)
        exit(errno);

    std::cout << "P3\n" << width << " " << height << "\n255\n";
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j * width + i;
            float r = fb[pixel_index].r();
            float g = fb[pixel_index].g();
            float b = fb[pixel_index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    fclose(fd);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));  
    
    hipDeviceReset();
}